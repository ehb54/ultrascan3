#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "us_cuda.h"

#if defined( USE_MPI )
#   include "us_semaphore.h"
    extern int myrank;;
    int env_mpi_node;
#else 
#   define us_semaphore_unlock(x) ()
#endif

static int semid;

#  define CUDA_SAFE_CALL_NO_SYNC( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        us_semaphore_unlock( semid );                                        \
        return false;                                                        \
    } }

#  define CUDA_SAFE_CALL( call)     CUDA_SAFE_CALL_NO_SYNC(call);                                            \

#  define CUT_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        us_semaphore_unlock( semid );                                        \
        return false;                                                        \
    }                                                                        \
    }

// Prototypes
__global__ void helloWorld( char * );
__global__ void cudaDebye( 
                          unsigned int  n,
                          unsigned int  q_points,
                          const float * q,
                          const float * pos,
                          const float * fp,
                          float *       I
                          );

#if defined( USE_MPI )
bool cuda_ipcrm()
{
  // printf( "%d:cuda_ipcrm called node %d\n", myrank, env_mpi_node ); 
   int deviceCount; 
   hipGetDeviceCount(&deviceCount); 
   for ( int i = 0; i < deviceCount; i++ )
   {
     int okid = myrank % deviceCount;
     key_t key = (key_t) ( 0x54000001 + env_mpi_node * 0x00001000 + okid );
     // printf( "%d:cuda_ipcrm called node %d key 0x%x device %d\n", myrank, env_mpi_node, key, okid ); 
     if ( !us_semaphore_exists( key, 1, semid ) )
     {
       // perror( "us_semaphore_exists");
       // printf( "%d:cuda_ipcrm non existant semaphore\n", myrank ); 
       return true;
     }
     if ( !us_semaphore_free( semid ) ) 
     {
       // perror( "us_semaphore_free");
       // printf( "%d:cuda_ipcrm error freeing semaphore\n", myrank ); 
       return false;
     }
   }
   // printf( "%d:cuda_ipcrm successfully removed\n", myrank ); 
   return true;
}
#endif      

// cuda debye, everything must be preallocated
bool cuda_debye( 
                bool         autocorrelate,
                unsigned int n,   // number of atoms
                float *      pos, // each atom will have 3 floats for coordinates, 
                // so this will have 3 * n entries

                unsigned int q_points,
                float *      q,   // this is the q grid

                float *      fp,  // this has the scattering factors for each atom
                // it is 2d and it will have n * q_points entries
                // currently it is in atom, q order
                // it would be good to test q, atom order since we are currently 
                // computing I(q) with q per thread
  
                // output:
                float *      I,    // the computed debye output, q_points
                unsigned int  threads_per_block
#if defined( USE_MPI )
		, int myrank
#endif
                )
{

#define MAXN 10000000

   if ( n > MAXN )
   {
      printf( "n was %u, now set to %u\n", n, MAXN );
      n = MAXN;
   }

   int deviceCount; 
   hipGetDeviceCount(&deviceCount); 

   hipDeviceProp_t props;
  
   int okid = -1;

   for ( int device = 0; device < deviceCount; ++device ) 
   { 
      CUDA_SAFE_CALL( hipGetDeviceProperties( &props, device ) );

      printf("Device %d: \"%s\" with Compute %d.%d capability\n", device, props.name, props.major, props.minor);
      printf( 
             "totalGlobalMem               %lu\n"
             "sharedMemPerBlock            %d\n"
             "regsPerBlock                 %d\n"
             "warpSize                     %d\n"
             "memPitch                     %d\n"
             "maxThreadsPerBlock           %d\n"
             "maxThreadsDim                %d %d %d\n"
             "maxGridSize                  %d %d %d\n"
             "clockRate                    %d\n"
             "totalConstMem                %d\n"
             "multiProcessorCount          %d\n"
             "kernelExecTimeoutEnabled     %d\n"
             "canMapHostMemory             %d\n"
             "computeMode                  %d\n"
             "concurrentKernels            %d\n"
             "asyncEngineCount             %d\n"
             "unifiedAddressing            %d\n"
             "memoryClockRate              %d\n"
             "memoryBusWidth               %d\n"
             "l2CacheSize                  %d\n"
             "maxThreadsPerMultiProcessor  %d\n"
             
             , props.totalGlobalMem
             , props.sharedMemPerBlock
             , props.regsPerBlock
             , props.warpSize
             , props.memPitch
             , props.maxThreadsPerBlock
             , props.maxThreadsDim[0]
             , props.maxThreadsDim[1]
             , props.maxThreadsDim[2]
             , props.maxGridSize  [0]
             , props.maxGridSize  [1]
             , props.maxGridSize  [2]
             , props.clockRate
             , props.totalConstMem
             , props.multiProcessorCount
             , props.kernelExecTimeoutEnabled
             , props.canMapHostMemory
             , props.computeMode
             , props.concurrentKernels
             , props.asyncEngineCount
             , props.unifiedAddressing
             , props.memoryClockRate
             , props.memoryBusWidth
             , props.l2CacheSize
             , props.maxThreadsPerMultiProcessor
             );
      if ( props.kernelExecTimeoutEnabled )
      {
         printf( "Warning: cuda kernel timeout enabled on device %d.  Disconnect the display and try again!\n", device );
      } else {
         okid = device;
      }
   }

#if defined( USE_MPI )
   // swap per rank
   okid = myrank % deviceCount;
   key_t key = (key_t) ( 0x54000001 + env_mpi_node * 0x00001000 + okid );
   if ( !us_semaphore_init( key, 1, semid ) )
   {
      perror( "us_semaphore_init");
      return false;
   }
   printf( "%d:cuda_lock called node %d key 0x%x device %d\n", myrank, env_mpi_node, key, okid ); 
   if ( !us_semaphore_lock( semid ) ) 
   {
      perror( "us_semaphore_lock");
      return false;
   }
#endif

   if ( okid != -1 )
   {
      printf( "setting to device %d\n", okid );
      CUDA_SAFE_CALL( hipSetDevice( okid ) );
   }

   // each thread will create its own I which we will have to sum at the end
   unsigned int threads         = q_points;
      
   unsigned int threadsPerBlock = threads_per_block;
   unsigned int blocksPerGrid   = (q_points + threadsPerBlock - 1) / threadsPerBlock;


   unsigned long memory_req     = 
      (unsigned long) ( q_points *     sizeof( float ) ) +
      (unsigned long) ( 3 * n *        sizeof( float ) ) +
      (unsigned long) ( n * q_points * sizeof( float ) ) +
      (unsigned long) ( q_points     * sizeof( float ) );;

   printf( "cuda_debye:\n"
           "memory required : %lu\n"
           "threads         : %u\n"
           "threadsperblock : %u\n"
           "blockspergrid   : %u\n"
           , memory_req
           , threads
           , threadsPerBlock
           , blocksPerGrid
           );

   if ( props.totalGlobalMem < memory_req )
   {
      printf( "insufficient memory to run cuda debye on this device" );
      return false;
   }

   printf( "starting cudaDebye\n" );

   // Allocate vectors in device memory
   float * d_pos;
   float * d_fp;
   float * d_q; 
   float * d_I; 

   CUDA_SAFE_CALL( hipMalloc( (void**)&d_q  , q_points *     sizeof( float ) ) );
   CUDA_SAFE_CALL( hipMalloc( (void**)&d_pos, 3 * n *        sizeof( float ) ) );
   CUDA_SAFE_CALL( hipMalloc( (void**)&d_fp , n * q_points * sizeof( float ) ) );
   CUDA_SAFE_CALL( hipMalloc( (void**)&d_I  , q_points     * sizeof( float ) ) );

   printf( "cudasync 0\n" );
   CUDA_SAFE_CALL( hipDeviceSynchronize() );

   // Copy vectors from host memory to device memory
   CUDA_SAFE_CALL( hipMemcpy( d_q  , q  , q_points *     sizeof( float ) , hipMemcpyHostToDevice) );
   CUDA_SAFE_CALL( hipMemcpy( d_pos, pos, 3 * n *        sizeof( float ) , hipMemcpyHostToDevice) );
   CUDA_SAFE_CALL( hipMemcpy( d_fp , fp , n * q_points * sizeof( float ) , hipMemcpyHostToDevice) );

   printf( "cudasync 1\n" );
   CUDA_SAFE_CALL( hipDeviceSynchronize() );

   // Invoke kernel
   cudaDebye<<<blocksPerGrid, threadsPerBlock>>>( n, q_points, d_q, d_pos, d_fp, d_I );

   CUT_CHECK_ERROR( "cudaDebye() execution failed\n" );

   // wait for kernel to finish
   printf( "cudasync 2\n" );
   hipDeviceSynchronize();


   // Copy result from device memory to host memory
   // h_C contains the result in host memory
   CUDA_SAFE_CALL( hipMemcpy( I, d_I, q_points * sizeof( float ), hipMemcpyDeviceToHost) );

   bool all_zero = true;
   unsigned no_of_nonzeros = 0;
   for ( unsigned int i = 0; i < q_points; i++ )
   {
      if ( I[ i ] != 0e0 )
      {
         all_zero = false;
         no_of_nonzeros++;
      }
   }

   if ( all_zero )
   {
      printf( "hmm: cudaDebye returned I of all zeros!\n" );
   } else {
      printf( "hmm: cudaDebye had %u of %u nonzero\n", no_of_nonzeros, q_points );
   }      

   if ( d_q )
   {
      hipFree( d_q );
   }
   if ( d_pos )
   {
      hipFree( d_pos );
   }
   if ( d_fp )
   {
      hipFree( d_fp );
   }
   if ( d_I )
   {
      hipFree( d_I );
   }

   printf( "cudasync 3\n" );
   CUDA_SAFE_CALL( hipDeviceSynchronize() );

#if defined( USE_MPI )
   if ( !us_semaphore_unlock( semid ) ) 
   {
      perror( "us_semaphore_lock");
      return false;
   }
#endif

   printf( "end cudaDebye\n" );
   return true;
}

// Host function
bool
cuda_hello_world()
{
   int deviceCount; 
   hipGetDeviceCount(&deviceCount); 

   hipDeviceProp_t props;

   int okid = -1;

   for ( int device = 0; device < deviceCount; ++device ) 
   { 
      CUDA_SAFE_CALL( hipGetDeviceProperties( &props, device ) );

      printf("Device %d: \"%s\" with Compute %d.%d capability\n", device, props.name, props.major, props.minor);
      printf( 
             "totalGlobalMem               %lu\n"
             "sharedMemPerBlock            %d\n"
             "regsPerBlock                 %d\n"
             "warpSize                     %d\n"
             "memPitch                     %d\n"
             "maxThreadsPerBlock           %d\n"
             "maxThreadsDim                %d %d %d\n"
             "maxGridSize                  %d %d %d\n"
             "clockRate                    %d\n"
             "totalConstMem                %d\n"
             "multiProcessorCount          %d\n"
             "kernelExecTimeoutEnabled     %d\n"
             "canMapHostMemory             %d\n"
             "computeMode                  %d\n"
             "concurrentKernels            %d\n"
             "asyncEngineCount             %d\n"
             "unifiedAddressing            %d\n"
             "memoryClockRate              %d\n"
             "memoryBusWidth               %d\n"
             "l2CacheSize                  %d\n"
             "maxThreadsPerMultiProcessor  %d\n"
             
             , props.totalGlobalMem
             , props.sharedMemPerBlock
             , props.regsPerBlock
             , props.warpSize
             , props.memPitch
             , props.maxThreadsPerBlock
             , props.maxThreadsDim[0]
             , props.maxThreadsDim[1]
             , props.maxThreadsDim[2]
             , props.maxGridSize  [0]
             , props.maxGridSize  [1]
             , props.maxGridSize  [2]
             , props.clockRate
             , props.totalConstMem
             , props.multiProcessorCount
             , props.kernelExecTimeoutEnabled
             , props.canMapHostMemory
             , props.computeMode
             , props.concurrentKernels
             , props.asyncEngineCount
             , props.unifiedAddressing
             , props.memoryClockRate
             , props.memoryBusWidth
             , props.l2CacheSize
             , props.maxThreadsPerMultiProcessor
             );
      
      if ( props.kernelExecTimeoutEnabled )
      {
         printf( "Warning: cuda kernel timeout enabled on device %d.  Disconnect the display and try again!\n", device );
      } else {
         okid = device;
      }
   }

   if ( okid != -1 )
   {
      printf( "setting to device %d\n", okid );
      CUDA_SAFE_CALL( hipSetDevice( okid ) );
   }

   int i;

   // desired output
   char str[] = "Hello World!";

   // mangle contents of output
   // the null character is left intact for simplicity
   for(i = 0; i < 12; i++)
      str[i] -= i;

   // allocate memory on the device 
   char *d_str;
   size_t size = sizeof(str);
   hipMalloc((void**)&d_str, size);

   // copy the string to the device
   hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

   // set the grid and block sizes
   dim3 dimGrid(2);   // one block per word  
   dim3 dimBlock(6); // one thread per character
  
   // invoke the kernel
   helloWorld<<< dimGrid, dimBlock >>>(d_str);

   // retrieve the results from the device
   CUDA_SAFE_CALL( hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost) );

   // free up the allocated memory on the device
   CUDA_SAFE_CALL ( hipFree(d_str) );
  
   // everyone's favorite part
   printf("%s\n", str);

   return strncmp( str, "Hello World!" , 12 );
}

bool
cuda_reset()
{
   CUDA_SAFE_CALL( hipDeviceReset        (               ) );
   return true;
}
   

// Device kernels
__global__ void

/*
** Hello World using CUDA
** 
** The string "Hello World!" is mangled then restored using a common CUDA idiom
**
** Byron Galbraith
** 2009-02-18
*/

helloWorld( char* str )
{
   // determine where in the thread grid we are
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

   // unmangle output
   str[idx] += idx;
}

// currently autocorrolate always on

__global__ void
cudaDebye( 
          unsigned int  n,
          unsigned int  q_points,
          const float * q,
          const float * pos,
          const float * fp,
          float *       I
          )
{
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   // all blocks are full of threads so we may have more than needed
   if ( idx < q_points )
   {
      I[ idx ] = 0.0;

      const unsigned int nm1 = n - 1;
      unsigned int       iofs;
      unsigned int       jofs;
      float              sqrikd;
      float              qrik;
      const unsigned int fpbase = idx * n;

      float              fpi;

      unsigned int       i;
      unsigned int       j;

      for ( i = 0; i < nm1; ++i )
      {
         iofs = i * 3;
         fpi  = fp[ fpbase + i ];

         for ( j = i + 1; j < n; ++j )
         {
            jofs = j * 3;
            qrik = 
               q[ idx ] *
               sqrt(
                    ( pos[ iofs ]     - pos[ jofs ]     ) * ( pos[ iofs ]     - pos[ jofs ]     ) +
                    ( pos[ iofs + 1 ] - pos[ jofs + 1 ] ) * ( pos[ iofs + 1 ] - pos[ jofs + 1 ] ) +
                    ( pos[ iofs + 2 ] - pos[ jofs + 2 ] ) * ( pos[ iofs + 2 ] - pos[ jofs + 2 ] ) 
                    );

            sqrikd = ( fabs( qrik ) < 1e-20 ) ? 1.0 : sin( qrik ) / qrik;

            I[ idx ] += 2.0 * sqrikd * fpi * fp[ fpbase + j ];
         }
         I[ idx ] += fpi * fpi;
      }
      I[ idx ] += fp[ fpbase + nm1 ] * fp[ fpbase + nm1 ];
   }
}
